#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include "utils.h"

#define N_THREADS 1024
#define N_BLOCKS 16

/*** GPU functions ***/
__global__ void init_rand_kernel(hiprandState *state) {
 int idx = blockIdx.x * blockDim.x + threadIdx.x;
 hiprand_init(0, idx, 0, &state[idx]);        
}
__global__ void random_walk_kernel(float *map, int rows, int cols, int* bx, int* by,
                                   int steps, hiprandState *state) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  float start = hiprand_uniform(&state[tid]);
  int idx = start * rows * cols;
  int x = idx / cols;
  int y = (idx - 1) % cols;

  float max_height = map[idx];
  bx[tid] = x;
  by[tid] = y;
  //printf("beggining height %f\n", max_height);

  for(int i = 0; i < steps; i++) {
    float randNum = hiprand_uniform(&state[tid]);
    int randIdx = int(4 * randNum);
    //printf("here %d\n", randNum);
    switch (randIdx) {
      case 0:
        if (y != 0) {
          y = y - 1;
        }
        break;
      case 1:
        if (x != 0) {
          x = x - 1;
        }
        break;
      case 2:
        if (y < 6113) {
        y = y + 1;
        break;
        }
      case 3:
      if (x < 2047) {
        x = x + 1;
        break;
      }
    }
    idx = x * 6114 + y;
    float height = map[idx];
    if (height > max_height) {
      max_height = height;
      bx[tid] = x;
      by[tid] = y;
    }
  }
 
  //TODO: implement random walk!
}

__global__ void local_max_kernel(float *map, int rows, int cols, int* bx, int* by,
                                 int steps, hiprandState *state) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  //TODO: implement local max!
}

__global__ void local_max_restart_kernel(float *map, int rows, int cols, int* bx,
                                         int* by, int steps, hiprandState *state) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  //TODO: implement local max with restarts!
}

/*** CPU functions ***/
hiprandState* init_rand() {
  hiprandState *d_state;
  hipMalloc(&d_state, N_BLOCKS * N_THREADS * sizeof(hiprandState));
  init_rand_kernel<<<N_BLOCKS, N_THREADS>>>(d_state);
  return d_state;
}


float random_walk(float* map, int rows, int cols, int steps) {
  hiprandState* d_state = init_rand();
  int *bx, *by;
  int *d_bx, *d_by;
  float* d_map;


  // Before kernel call:
  // Need to allocate memory for above variables and copy data to GPU

  
  

  bx = (int*)malloc(N_BLOCKS * N_THREADS * sizeof(float));
  by = (int*)malloc(N_BLOCKS * N_THREADS * sizeof(float));

  hipMalloc(&d_bx, N_BLOCKS * N_THREADS * sizeof(float));
  hipMalloc(&d_by, N_BLOCKS * N_THREADS * sizeof(float));

  

  hipMalloc(&d_map, rows * cols * sizeof(float));
  hipMemcpy(d_map, map, rows * cols * sizeof(float), hipMemcpyHostToDevice);

  random_walk_kernel<<<N_BLOCKS, N_THREADS>>>(d_map, rows, cols, d_bx, d_by, steps, d_state);

  hipMemcpy(map, d_map, N_BLOCKS * N_THREADS * sizeof(float), hipMemcpyDeviceToHost);

  hipMemcpy(bx, d_bx, N_BLOCKS * N_THREADS * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(by, d_by, N_BLOCKS * N_THREADS * sizeof(float), hipMemcpyDeviceToHost);

  // After kernel call:
  // Need to copy data back to CPU and find max value
  float max_val = 0;
  for(int i = 0; i < N_BLOCKS * N_THREADS; i++) {
    int x = bx[i];
    int y = by[i];
    int idx = x * 6114 + y + 1;
    float height = map[idx];
    if (height > max_val) {
      max_val = height;
    }
  }

  // Finally: free used GPU and CPU memory
  //hipFree(d_state);
  //hipFree(d_map);
  //free(map);
  return max_val;
}

// Work on these after finishing random walk
float local_max(float* map, int rows, int cols, int steps);
float local_max_restart(float* map, int rows, int cols, int steps);


int main(int argc, char** argv) {
  if (argc != 2) {
    printf("Usage: %s <map_file> \n", argv[0]);
    return 1;
  }

  float *map;
  int rows, cols;
  read_bin(argv[1], &map, &rows, &cols);
  printf("%d %d\n", rows, cols);


  // As a starting point, try to get it working with a single steps value
  int steps = 1028;
  float max_val = random_walk(map, rows, cols, steps);
  
  printf("%d %d\n", rows, cols);
  printf("Random walk max value: %f\n", max_val);

  return 0;
}