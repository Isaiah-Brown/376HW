#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include "utils.h"

#define N_THREADS 1
#define N_BLOCKS 1

/*** GPU functions ***/
__global__ void init_rand_kernel(hiprandState *state) {
 int idx = blockIdx.x * blockDim.x + threadIdx.x;
 hiprand_init(0, idx, 0, &state[idx]);        
}
__global__ void random_walk_kernel(float *map, int rows, int cols, int* bx, int* by,
                                   int steps, hiprandState *state) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  printf("%d\n", tid);
  //TODO: implement random walk!
}

__global__ void local_max_kernel(float *map, int rows, int cols, int* bx, int* by,
                                 int steps, hiprandState *state) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  //TODO: implement local max!
}

__global__ void local_max_restart_kernel(float *map, int rows, int cols, int* bx,
                                         int* by, int steps, hiprandState *state) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  //TODO: implement local max with restarts!
}

/*** CPU functions ***/
hiprandState* init_rand() {
  hiprandState *d_state;
  hipMalloc(&d_state, N_BLOCKS * N_THREADS * sizeof(hiprandState));
  init_rand_kernel<<<N_BLOCKS, N_THREADS>>>(d_state);
  return d_state;
}


float random_walk(float* map, int rows, int cols, int steps) {
  hiprandState* d_state = init_rand();
  int *bx, *by;
  int *d_bx, *d_by;
  float* d_map;

  // Before kernel call:
  // Need to allocate memory for above variables and copy data to GPU

  /*
  float max = 0;
  float min = 0;
  for(int i = 0; i < rows * cols; i++) {
      if (map[i] > max) {
        max = map[i];
        printf("%f\n", max);
      }
      if (map[i] < min) {
        min = map[i];
        printf("%f\n", min);
      }
      if( i % 2048 == 0) {
        printf("%d\n", i);
      }
      
  }
  */

  bx = (int*)malloc(N_BLOCKS * N_THREADS * sizeof(float));
  by = (int*)malloc(N_BLOCKS * N_THREADS * sizeof(float));

  hipMalloc(&d_bx, N_BLOCKS * N_THREADS * sizeof(float));
  hipMalloc(&d_by, N_BLOCKS * N_THREADS * sizeof(float));

  

  hipMalloc(&d_map, rows * cols * sizeof(float));
  hipMemcpy(d_map, map, rows * cols * sizeof(float), hipMemcpyHostToDevice);

  random_walk_kernel<<<N_BLOCKS, N_THREADS>>>(d_map, rows, cols, d_bx, d_by, steps, d_state);

  hipMemcpy(map, d_map, N_BLOCKS * N_THREADS * sizeof(float), hipMemcpyDeviceToHost);

  hipMemcpy(bx, d_bx, N_BLOCKS * N_THREADS * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(by, d_by, N_BLOCKS * N_THREADS * sizeof(float), hipMemcpyDeviceToHost);

  // After kernel call:
  // Need to copy data back to CPU and find max value

  

  float max_val = 0;

  // Finally: free used GPU and CPU memory
  //hipFree(d_state);
  //hipFree(d_map);
  //free(map);
  return max_val;
}

// Work on these after finishing random walk
float local_max(float* map, int rows, int cols, int steps);
float local_max_restart(float* map, int rows, int cols, int steps);


int main(int argc, char** argv) {
  if (argc != 2) {
    printf("Usage: %s <map_file> \n", argv[0]);
    return 1;
  }

  float *map;
  int rows, cols;
  read_bin(argv[1], &map, &rows, &cols);
  printf("%d %d\n", rows, cols);


  // As a starting point, try to get it working with a single steps value
  int steps = 10;
  float max_val = random_walk(map, rows, cols, steps);
  
  printf("%d %d\n", rows, cols);
  printf("Random walk max value: %f\n", max_val);

  return 0;
}