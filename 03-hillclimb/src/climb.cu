#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include "utils.h"

#define N_THREADS 1024
#define N_BLOCKS 16

/*** GPU functions ***/
__global__ void init_rand_kernel(hiprandState *state) {
 int idx = blockIdx.x * blockDim.x + threadIdx.x;
 hiprand_init(0, idx, 0, &state[idx]);        
}
__global__ void random_walk_kernel(float *map, int rows, int cols, int* bx, int* by,
                                   int steps, hiprandState *state) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  float start = hiprand_uniform(&state[tid]);
  int idx = start * rows * cols;
  int x = idx / cols; 
  int y = idx % cols;

  float max_height = map[idx];
  bx[tid] = x;
  by[tid] = y;

  for(int i = 0; i < steps; i++) {
    float randNum = hiprand_uniform(&state[tid]);
    int randIdx = int(4 * randNum); //make number from 0-3
  
    switch (randIdx) {
      case 0:
        if (y != 0) { //left
          y = y - 1;
        } else if (x != 0) { //if y = 0 than decrease x
          x = x - 1;
        }
        break;
      case 1:
        if (x != 0) {  //top
          x = x - 1;
        } else if (y != 0) { // if x= 0 than decrease y
            y = y - 1;
        }
        break;
      case 2:
        if (y != (cols - 1)) { //right
          y = y + 1;
        } else if (x != (rows - 1)) { //if y is in last column increase x
          x = x + 1;
        }
        break;
      case 3:
      if (x != (rows - 1)) { //bottom
        x = x + 1;
        break;
      } else if (y != (cols - 1)) { //if x is in last row increase y
            y = y + 1;
        break;
      }
    }
    idx = x * 6114 + y; //make single index
    float height = map[idx];
    if (height > max_height) {
      max_height = height;
      bx[tid] = x;
      by[tid] = y;
    }
  }
 
  //TODO: implement random walk!
}

__global__ void local_max_kernel(float *map, int rows, int cols, int* bx, int* by,
                                 int steps, hiprandState *state) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  float start = hiprand_uniform(&state[tid]);
  int idx = start * rows * cols;
  int x = idx / cols;
  int y = idx % cols;

  float max_height = map[idx];
  bx[tid] = x;
  by[tid] = y;

  for(int i = 0; i < steps; i++) {

    float left, top, right, bottom; //initalize floats for the pixels to the left, right, etc of the current pixel

    if (y != 0) { //y = y - 1;
      int leftIdx = (x * cols) + y - 1;
      left = map[leftIdx];
    } else {
      left = -99;
    }

    if (x != 0) {   //x = x - 1;
      int topIdx = ((x - 1) * cols) + y;
      top = map[topIdx];
    } else {
      top = -99;
    }
       
     
    if (y != (cols -1)) {  //y = y + 1;
      int rightIdx = (x * cols) + y + 1;
      right = map[rightIdx];
      
    } else {
      right = -99;
    }
    
    if (x != (rows - 1)) {  //x = x + 1;
      int bottomIdx = ((x + 1) * cols) + y;
      bottom = map[bottomIdx];
    } else {
      bottom = -99;
    }

    if (left > top  && left > right && left > bottom) { //find highest of left, right etc
      y = y - 1;
    } else if (top > left && top > right && top > bottom) {
      x = x - 1;
    } else if (right > left && right > top && right > bottom) {
      y = y + 1;
    } else {
      x = x + 1;
    }

    int currMaxIdx = x * 6114 + y; 
    float currMax = map[currMaxIdx]; //map value at the best index
    
    if (currMax > max_height) {
      max_height = currMax;
      bx[tid] = x;
      by[tid] = y;
    }
  }
 
  //TODO: implement local max!
}

__global__ void local_max_restart_kernel(float *map, int rows, int cols, int* bx,
                                         int* by, int steps, hiprandState *state) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  float start = hiprand_uniform(&state[tid]);
  int idx = start * rows * cols;
  int x = idx / cols;
  int y = idx % cols;

  float max_height = map[idx];
  bx[tid] = x;
  by[tid] = y;
  
  int last_max = 0;
  for(int i = 0; i < steps; i++) {

    float left, top, right, bottom;

    
    if (y != 0) {
      int leftIdx = (x * cols) + y - 1;
      left = map[leftIdx];
      //y = y - 1;
    } else {
      left = -99;
    }

    if (x != 0) {
      int topIdx = ((x - 1) * cols) + y;
      top = map[topIdx];
      //x = x - 1;
    } else {
      top = -99;
    }
       
     
    if (y != (cols -1)) {
      int rightIdx = (x * cols) + y + 1;
      right = map[rightIdx];
      //y = y + 1;
      
    } else {
      right = -99;
    }
    
    if (x != (rows - 1)) {
      int bottomIdx = ((x + 1) * cols) + y;
      bottom = map[bottomIdx];
      //x = x + 1;
    } else {
      bottom = -99;
    }

    if (left > top  && left > right && left > bottom) {
      y = y - 1;
    } else if (top > left && top > right && top > bottom) {
      x = x - 1;
    } else if (right > left && right > top && right > bottom) {
      y = y + 1;
    } else {
      x = x + 1;
    }

    int currMaxIdx = x * 6114 + y;
    float currMax = map[currMaxIdx];
    last_max = last_max + 1;    //add a step since last max
    
    if (currMax > max_height) {
      last_max = 0;
      max_height = currMax;
      bx[tid] = x;
      by[tid] = y;
    }

    if (last_max > 4) { //get a new index
      float newStart = hiprand_uniform(&state[tid]);
      int newIdx = newStart * rows * cols;
      x = newIdx / cols;
      y = newIdx % cols;
    }

  }
  //TODO: implement local max with restarts!
}

/*** CPU functions ***/
hiprandState* init_rand() {
  hiprandState *d_state;
  hipMalloc(&d_state, N_BLOCKS * N_THREADS * sizeof(hiprandState));
  init_rand_kernel<<<N_BLOCKS, N_THREADS>>>(d_state);
  return d_state;
}


float random_walk(float* map, int rows, int cols, int steps) {
  hiprandState* d_state = init_rand();
  int *bx, *by;
  int *d_bx, *d_by;
  float* d_map;

  // Before kernel call:
  // Need to allocate memory for above variables and copy data to GPU

  bx = (int*)malloc(N_BLOCKS * N_THREADS * sizeof(float));
  by = (int*)malloc(N_BLOCKS * N_THREADS * sizeof(float));
  hipMalloc(&d_bx, N_BLOCKS * N_THREADS * sizeof(float));
  hipMalloc(&d_by, N_BLOCKS * N_THREADS * sizeof(float));

  
  hipMalloc(&d_map, rows * cols * sizeof(float));
  hipMemcpy(d_map, map, rows * cols * sizeof(float), hipMemcpyHostToDevice);

  random_walk_kernel<<<N_BLOCKS, N_THREADS>>>(d_map, rows, cols, d_bx, d_by, steps, d_state);

  hipMemcpy(map, d_map, N_BLOCKS * N_THREADS * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(bx, d_bx, N_BLOCKS * N_THREADS * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(by, d_by, N_BLOCKS * N_THREADS * sizeof(float), hipMemcpyDeviceToHost);

  // After kernel call:
  // Need to copy data back to CPU and find max value
  float max_val = 0;
  for(int i = 0; i < N_BLOCKS * N_THREADS; i++) {
    int x = bx[i];
    int y = by[i];
    int idx = x * 6114 + y;
    float height = map[idx];
    if (height > max_val) {
      max_val = height;
    }
  }

  // Finally: free used GPU and CPU memory
  hipFree(d_state);
  hipFree(d_map);
  hipFree(d_bx);
  hipFree(d_by);
  free(bx);
  free(by);
  return max_val;
}

// Work on these after finishing random walkif (y != 0)
float local_max(float* map, int rows, int cols, int steps){
  hiprandState* d_state = init_rand();
  int *bx, *by;
  int *d_bx, *d_by;
  float* d_map;

  // Before kernel call:
  // Need to allocate memory for above variables and copy data to GPU

  bx = (int*)malloc(N_BLOCKS * N_THREADS * sizeof(float));
  by = (int*)malloc(N_BLOCKS * N_THREADS * sizeof(float));
  hipMalloc(&d_bx, N_BLOCKS * N_THREADS * sizeof(float));
  hipMalloc(&d_by, N_BLOCKS * N_THREADS * sizeof(float));

  
  hipMalloc(&d_map, rows * cols * sizeof(float));
  hipMemcpy(d_map, map, rows * cols * sizeof(float), hipMemcpyHostToDevice);

  local_max_kernel<<<N_BLOCKS, N_THREADS>>>(d_map, rows, cols, d_bx, d_by, steps, d_state);

  hipMemcpy(map, d_map, N_BLOCKS * N_THREADS * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(bx, d_bx, N_BLOCKS * N_THREADS * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(by, d_by, N_BLOCKS * N_THREADS * sizeof(float), hipMemcpyDeviceToHost);

  // After kernel call:
  // Need to copy data back to CPU and find max value
  float max_val = 0;
  for(int i = 0; i < N_BLOCKS * N_THREADS; i++) {
    int x = bx[i];
    int y = by[i];
    int idx = x * 6114 + y;
    float height = map[idx];
    if (height > max_val) {
      max_val = height;
    }
  }

  // Finally: free used GPU and CPU memory
  hipFree(d_state);
  hipFree(d_map);
  hipFree(d_bx);
  hipFree(d_by);
  free(bx);
  free(by);
  return max_val;
}
float local_max_restart(float* map, int rows, int cols, int steps) {
  hiprandState* d_state = init_rand();
  int *bx, *by;
  int *d_bx, *d_by;
  float* d_map;

  // Before kernel call:
  // Need to allocate memory for above variables and copy data to GPU

  bx = (int*)malloc(N_BLOCKS * N_THREADS * sizeof(float));
  by = (int*)malloc(N_BLOCKS * N_THREADS * sizeof(float));
  hipMalloc(&d_bx, N_BLOCKS * N_THREADS * sizeof(float));
  hipMalloc(&d_by, N_BLOCKS * N_THREADS * sizeof(float));

  
  hipMalloc(&d_map, rows * cols * sizeof(float));
  hipMemcpy(d_map, map, rows * cols * sizeof(float), hipMemcpyHostToDevice);

  local_max_restart_kernel<<<N_BLOCKS, N_THREADS>>>(d_map, rows, cols, d_bx, d_by, steps, d_state);

  hipMemcpy(map, d_map, N_BLOCKS * N_THREADS * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(bx, d_bx, N_BLOCKS * N_THREADS * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(by, d_by, N_BLOCKS * N_THREADS * sizeof(float), hipMemcpyDeviceToHost);

  // After kernel call:
  // Need to copy data back to CPU and find max value
  float max_val = 0;
  for(int i = 0; i < N_BLOCKS * N_THREADS; i++) {
    int x = bx[i];
    int y = by[i];
    int idx = x * 6114 + y;
    float height = map[idx];
    if (height > max_val) {
      max_val = height;
    }
  }

  // Finally: free used GPU and CPU memory
  hipFree(d_state);
  hipFree(d_map);
  hipFree(d_bx);
  hipFree(d_by);
  free(bx);
  free(by);
  return max_val;
}


int main(int argc, char** argv) {
  if (argc != 2) {
    printf("Usage: %s <map_file> \n", argv[0]);
    return 1;
  }

  float *map;
  int rows, cols;
  read_bin(argv[1], &map, &rows, &cols);
  printf("%d %d\n", rows, cols);


  // As a starting point, try to get it working with a single steps value
  int steps = 1;
  while(steps <= 1024) {
    float max_val = local_max_restart(map, rows, cols, steps);
    //printf("%d %d\n", rows, cols);
    printf("with reset: %d, max value: %f\n", steps, max_val);
    steps = steps * 2;
  }
  free(map);

  return 0;
}