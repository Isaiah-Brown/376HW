#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include "utils.h"

#define N_THREADS 1024
#define N_BLOCKS 16

/*** GPU functions ***/
__global__ void init_rand_kernel(hiprandState *state) {
 int idx = blockIdx.x * blockDim.x + threadIdx.x;
 hiprand_init(0, idx, 0, &state[idx]);        
}
__global__ void random_walk_kernel(float *map, int rows, int cols, int* bx, int* by,
                                   int steps, hiprandState *state) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  float start = hiprand_uniform(&state[tid]);
  int idx = start * rows * cols;
  int x = idx / cols;
  int y = idx % cols;

  float max_height = map[idx];
  bx[tid] = x;
  by[tid] = y;

  for(int i = 0; i < steps; i++) {
    float randNum = hiprand_uniform(&state[tid]);
    int randIdx = int(4 * randNum);
  
    switch (randIdx) {
      case 0:
        if (y != 0) { //left
          y = y - 1;
        } else if (x != 0) {
          x = x - 1;
        }
        break;
      case 1:
        if (x != 0) {  //top
          x = x - 1;
        } else if (y != 0) {
            y = y - 1;
        }
        break;
      case 2:
        if (y != (cols - 1)) { //right
          y = y + 1;
        } else if (x != (rows - 1)) {
          x = x + 1;
        }
        break;
      case 3:
      if (x != (rows - 1)) { //bottom
        x = x + 1;
        break;
      } else if (y != (cols - 1)) {
            y = y + 1;
        break;
      }
    }
    idx = x * 6114 + y;
    float height = map[idx];
    if (height > max_height) {
      max_height = height;
      bx[tid] = x;
      by[tid] = y;
    }
  }
 
  //TODO: implement random walk!
}

__global__ void local_max_kernel(float *map, int rows, int cols, int* bx, int* by,
                                 int steps, hiprandState *state) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  float start = hiprand_uniform(&state[tid]);
  int idx = start * rows * cols;
  int x = idx / cols;
  int y = (idx - 1) % cols;

  float max_height = map[idx];
  bx[tid] = x;
  by[tid] = y;

  printf("%f\n", max_height);
  for(int i = 0; i < steps; i++) {
    //float randNum = hiprand_uniform(&state[tid]);
    //int randIdx = int(4 * randNum);

    float left, top, right, bottom;

    
    if (y != 0) {
      int leftIdx = x * 6114 + y - 1;
      left = map[leftIdx];
      //y = y - 1;
    } else {
      left = -99;
    }

    if (x != 0) {
      int topIdx = ((x - 1) * 6114) + y;
      top = map[topIdx];
      //x = x - 1;
    } else {
      top = -99;
    }
       
     
    if (y < 6113) {
      int rightIdx = x * 6114 + y + 1;
      top = map[rightIdx];
      //y = y + 1;
      
    } else {
      right = -99;
    }
    
    if (x < 2047) {
      int bottomIdx = ((x + 1) * 6114) + y - 1;
      top = map[bottomIdx];
      //x = x + 1;
    } else {
      bottom = -99;
    }

    if (left > top  && left > right && left > bottom) {
      y = y - 1;
    } else if (top > left && top > right && top > bottom) {
      x = x - 1;
    } else if (right > left && right > top && right > bottom) {
      y = y + 1;
    } else {
      x = x + 1;
    }

    int currMaxIdx = x * 6114 + y;
    float currMax = map[currMaxIdx];
    
    if (currMax > max_height) {
      max_height = currMax;
      bx[tid] = x;
      by[tid] = y;
      printf("%f\n", max_height);
    }
  }
 
  //TODO: implement local max!
}

__global__ void local_max_restart_kernel(float *map, int rows, int cols, int* bx,
                                         int* by, int steps, hiprandState *state) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  //TODO: implement local max with restarts!
}

/*** CPU functions ***/
hiprandState* init_rand() {
  hiprandState *d_state;
  hipMalloc(&d_state, N_BLOCKS * N_THREADS * sizeof(hiprandState));
  init_rand_kernel<<<N_BLOCKS, N_THREADS>>>(d_state);
  return d_state;
}


float random_walk(float* map, int rows, int cols, int steps) {
  hiprandState* d_state = init_rand();
  int *bx, *by;
  int *d_bx, *d_by;
  float* d_map;

  // Before kernel call:
  // Need to allocate memory for above variables and copy data to GPU

  bx = (int*)malloc(N_BLOCKS * N_THREADS * sizeof(float));
  by = (int*)malloc(N_BLOCKS * N_THREADS * sizeof(float));
  hipMalloc(&d_bx, N_BLOCKS * N_THREADS * sizeof(float));
  hipMalloc(&d_by, N_BLOCKS * N_THREADS * sizeof(float));

  
  hipMalloc(&d_map, rows * cols * sizeof(float));
  hipMemcpy(d_map, map, rows * cols * sizeof(float), hipMemcpyHostToDevice);

  random_walk_kernel<<<N_BLOCKS, N_THREADS>>>(d_map, rows, cols, d_bx, d_by, steps, d_state);

  hipMemcpy(map, d_map, N_BLOCKS * N_THREADS * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(bx, d_bx, N_BLOCKS * N_THREADS * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(by, d_by, N_BLOCKS * N_THREADS * sizeof(float), hipMemcpyDeviceToHost);

  // After kernel call:
  // Need to copy data back to CPU and find max value
  float max_val = 0;
  for(int i = 0; i < N_BLOCKS * N_THREADS; i++) {
    int x = bx[i];
    int y = by[i];
    int idx = x * 6114 + y;
    float height = map[idx];
    if (height > max_val) {
      max_val = height;
    }
  }

  // Finally: free used GPU and CPU memory
  hipFree(d_state);
  hipFree(d_map);
  hipFree(d_bx);
  hipFree(d_by);
  free(bx);
  free(by);
  return max_val;
}

// Work on these after finishing random walkif (y != 0)
float local_max(float* map, int rows, int cols, int steps){
  hiprandState* d_state = init_rand();
  int *bx, *by;
  int *d_bx, *d_by;
  float* d_map;

  // Before kernel call:
  // Need to allocate memory for above variables and copy data to GPU

  bx = (int*)malloc(N_BLOCKS * N_THREADS * sizeof(float));
  by = (int*)malloc(N_BLOCKS * N_THREADS * sizeof(float));
  hipMalloc(&d_bx, N_BLOCKS * N_THREADS * sizeof(float));
  hipMalloc(&d_by, N_BLOCKS * N_THREADS * sizeof(float));

  
  hipMalloc(&d_map, rows * cols * sizeof(float));
  hipMemcpy(d_map, map, rows * cols * sizeof(float), hipMemcpyHostToDevice);

  local_max_kernel<<<N_BLOCKS, N_THREADS>>>(d_map, rows, cols, d_bx, d_by, steps, d_state);

  hipMemcpy(map, d_map, N_BLOCKS * N_THREADS * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(bx, d_bx, N_BLOCKS * N_THREADS * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(by, d_by, N_BLOCKS * N_THREADS * sizeof(float), hipMemcpyDeviceToHost);

  // After kernel call:
  // Need to copy data back to CPU and find max value
  float max_val = 0;
  for(int i = 0; i < N_BLOCKS * N_THREADS; i++) {
    int x = bx[i];
    int y = by[i];
    int idx = x * 6114 + y + 1;
    float height = map[idx];
    if (height > max_val) {
      max_val = height;
    }
  }

  // Finally: free used GPU and CPU memory
  //hipFree(d_state);
  //hipFree(d_map);
  //hipFree(d_bx);
  //hipFree(d_by);
  //free(bx);
  //free(by);
  return max_val;
}
float local_max_restart(float* map, int rows, int cols, int steps);


int main(int argc, char** argv) {
  if (argc != 2) {
    printf("Usage: %s <map_file> \n", argv[0]);
    return 1;
  }

  float *map;
  int rows, cols;
  read_bin(argv[1], &map, &rows, &cols);
  printf("%d %d\n", rows, cols);


  // As a starting point, try to get it working with a single steps value
  int steps = 1;
  while(steps <= 1024) {
    float max_val = random_walk(map, rows, cols, steps);
    //printf("%d %d\n", rows, cols);
    printf("Local Max steps: %d, max value: %f\n", steps, max_val);
    steps = steps * 2;
  }
  free(map);

  return 0;
}